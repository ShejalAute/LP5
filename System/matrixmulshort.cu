
#include <iostream>
#include <hip/hip_runtime.h>

#define row1 2 /* Number of rows of first matrix */
#define col1 3 /* Number of columns of first matrix */
#define row2 3 /* Number of rows of second matrix */
#define col2 2 /* Number of columns of second matrix */

__global__ void matproductsharedmemory(int *l, int *m, int *n)
{
    int x = blockIdx.x;
    int y = blockIdx.y;

    // Calculate the index of the element to be computed
    int idx = y * col2 + x;

    // Initialize the value to store the product of the corresponding row of 'l' and column of 'm'
    int sum = 0;

    // Iterate over each element of the row of 'l' and column of 'm' to compute the dot product
    for (int i = 0; i < col1; ++i)
    {
        sum += l[y * col1 + i] * m[i * col2 + x];
    }

    // Store the result in the output matrix
    n[idx] = sum;
}

int main()
{
    int a[row1][col1];
    int b[row2][col2];
    int c[row1][col2];
    int *d, *e, *f;
    int i, j;

    std::cout << "\nEnter elements of first matrix of size 2*3\n";
    for (i = 0; i < row1; i++)
    {
        for (j = 0; j < col1; j++)
        {
            std::cin >> a[i][j];
        }
    }
    std::cout << "\nEnter elements of second matrix of size 3*2\n";
    for (i = 0; i < row2; i++)
    {
        for (j = 0; j < col2; j++)
        {
            std::cin >> b[i][j];
        }
    }

    hipMalloc((void **)&d, row1 * col1 * sizeof(int));
    hipMalloc((void **)&e, row2 * col2 * sizeof(int));
    hipMalloc((void **)&f, row1 * col2 * sizeof(int));

    hipMemcpy(d, a, row1 * col1 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(e, b, row2 * col2 * sizeof(int), hipMemcpyHostToDevice);

    dim3 grid(col2, row1);

    matproductsharedmemory<<<grid, 1>>>(d, e, f);

    hipMemcpy(c, f, row1 * col2 * sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "\nProduct of two matrices:\n";
    for (i = 0; i < row1; i++)
    {
        for (j = 0; j < col2; j++)
        {
            std::cout << c[i][j] << "\t";
        }
        std::cout << std::endl;
    }

    hipFree(d);
    hipFree(e);
    hipFree(f);

    return 0;
}


