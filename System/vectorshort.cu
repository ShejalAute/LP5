

#include <hip/hip_runtime.h>
#include <iostream>

__global__ void arradd(int *x, int *y, int *z) {
    int id = blockIdx.x;
    z[id] = x[id] + y[id];
}

int main() {
    int a[6];
    int b[6];
    int c[6];
    int *d, *e, *f;
    int i;

    std::cout << "Enter six elements of first array" << std::endl;
    for (i = 0; i < 6; i++) {
        std::cin >> a[i];
    }

    std::cout << "Enter six elements of second array" << std::endl;
    for (i = 0; i < 6; i++) {
        std::cin >> b[i];
    }

    hipMalloc((void **)&d, 6 * sizeof(int));
    hipMalloc((void **)&e, 6 * sizeof(int));
    hipMalloc((void **)&f, 6 * sizeof(int));

    hipMemcpy(d, a, 6 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(e, b, 6 * sizeof(int), hipMemcpyHostToDevice);

    arradd<<<6, 1>>>(d, e, f);

    hipMemcpy(c, f, 6 * sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "Sum of two arrays:" << std::endl;
    for (i = 0; i < 6; i++) {
        std::cout << c[i] << "\t";
    }
    std::cout << std::endl;

    hipFree(d);
    hipFree(e);
    hipFree(f);

    return 0;
}
